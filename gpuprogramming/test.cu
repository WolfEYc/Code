#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <chrono>
using namespace std;
using namespace std::chrono;

__global__
void add(int* arr, int n){
    int index = threadIdx.x;
    int stride = blockDim.x;
    for(int i = index; i < n; i+=stride){
        arr[i] = arr[i]*arr[i];
    }
}

void cpuadd(int* arr, int n){
    for(int i = 0; i < n; i++){
        arr[i] = arr[i]*arr[i];
    }
}

int main(void){

    int n;
    cin >> n;

    int* arr;
    int* arrcopy = new int [n];

    hipMallocManaged(& arr, n*sizeof(int));

    for(int i = 0; i < n; i++){
        arr[i] = i;
        arrcopy[i] = i;
    }

    auto gpustart = high_resolution_clock::now();

    add <<<1, 256>>> (arr,n);

    hipDeviceSynchronize();

    auto gpuend = high_resolution_clock::now();

    auto gputime = duration_cast<microseconds>(gpuend - gpustart);

    cout << "Gpu time: " << gputime.count() << endl;

    auto normalstart = high_resolution_clock::now();

    cpuadd(arrcopy,n);

    auto normalend = high_resolution_clock::now();

    auto normaltime = duration_cast<microseconds>(normalend - normalstart);

    cout<< "Normal time: " << normaltime.count() << endl;

    for(int i = 0; i < n; i++){
        if(arr[i] != arrcopy[i]){
            cout << "Arrays not equal" << endl;
            break;
        }
    }

    hipFree(arr);

    delete[] arrcopy;

    return 0;
}